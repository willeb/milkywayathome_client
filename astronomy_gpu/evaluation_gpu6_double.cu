#include "hip/hip_runtime.h"
/*
Copyright 2008, 2009 Travis Desell, Dave Przybylo, Nathan Cole,
Boleslaw Szymanski, Heidi Newberg, Carlos Varela, Malik Magdon-Ismail
and Rensselaer Polytechnic Institute.

This file is part of Milkway@Home.

Milkyway@Home is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

Milkyway@Home is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with Milkyway@Home.  If not, see <http://www.gnu.org/licenses/>.
*/

texture<int2, 3, hipReadModeElementType> tex_device_lb;
texture<int2, 2, hipReadModeElementType> tex_fstream_a;
texture<int2, 2, hipReadModeElementType> tex_fstream_c;
texture<int2, 2, hipReadModeElementType> tex_fstream_sigma_sq2;

hipArray **cu_arrays;

static __inline__ __device__
double tex3D_double(texture<int2, 3> tex, int x, int y, int z)
{
  int2 val = tex3D(tex, x, y, z);
  return __hiloint2double(val.x, val.y);
}

static __inline__ __device__
double tex2D_double(texture<int2, 2> tex, int x, int y)
{
  int2 val = tex2D(tex, x, y);
  return __hiloint2double(val.x, val.y);
}

//used to get the hi/lo parts of a double
//from NVIDIA CUDA SDK
volatile union cvt {
  double d;
  signed int i[2];
};

//converts an array of doubles
//into an array of int2s for the use
//in textures
int2*
convert(double* data, int size)
{
  int2 *ret = (int2 *) malloc(sizeof(int2) * size);
  for(unsigned int idx = 0;
      idx < size;++idx)
    {
      cvt cvt;
      cvt.d = data[idx];
      ret[idx].x = cvt.i[1];
      ret[idx].y = cvt.i[0];
    }
  return ret;
}

/**
   Similar to setup_texture except it deals with 2d textures
   that were previously in constant memory
 */
void setup_constant_textures(double *fstream_a, double *fstream_c, 
			     double *fstream_sigma_sq2, int number_streams)
{
  // allocate array
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int2>();
  hipArray* cu_array_a;
  cutilSafeCall(hipMallocArray(&cu_array_a, &channelDesc, 3, number_streams)); 
  cutilSafeCall(hipMemcpyToArray(cu_array_a, 0, 0, fstream_a, 3*number_streams, hipMemcpyHostToDevice));

  hipArray* cu_array_c;
  cutilSafeCall(hipMallocArray(&cu_array_c, &channelDesc, 3, number_streams)); 
  cutilSafeCall(hipMemcpyToArray(cu_array_c, 0, 0, fstream_c, 3*number_streams, hipMemcpyHostToDevice));

  hipArray* cu_array_sq2;
  cutilSafeCall(hipMallocArray(&cu_array_sq2, &channelDesc, 2, number_streams)); 
  cutilSafeCall(hipMemcpyToArray(cu_array_sq2, 0, 0, fstream_a, 2*number_streams, hipMemcpyHostToDevice));
  
  // set texture parameters
  tex_fstream_a.addressMode[0] = hipAddressModeClamp;
  tex_fstream_a.addressMode[1] = hipAddressModeClamp;
  tex_fstream_a.filterMode = hipFilterModePoint;
  tex_fstream_a.normalized = false;

  tex_fstream_c.addressMode[0] = hipAddressModeClamp;
  tex_fstream_c.addressMode[1] = hipAddressModeClamp;
  tex_fstream_c.filterMode = hipFilterModePoint;
  tex_fstream_c.normalized = false;

  tex_fstream_sigma_sq2.addressMode[0] = hipAddressModeClamp;
  tex_fstream_sigma_sq2.addressMode[1] = hipAddressModeClamp;
  tex_fstream_sigma_sq2.filterMode = hipFilterModePoint;
  tex_fstream_sigma_sq2.normalized = false;
  
  // Bind the array to the texture
  cutilSafeCall(hipBindTextureToArray(tex_fstream_a, cu_array_a, channelDesc));
  cutilSafeCall(hipBindTextureToArray(tex_fstream_c, cu_array_c, channelDesc));
  cutilSafeCall(hipBindTextureToArray(tex_fstream_sigma_sq2, cu_array_sq2, channelDesc));
}

/**
   Sets up a 3D texture for device__lb, right now it only
   support 1 integral.  In the future it should support
   more by copying from device memory the next integral.
 */
void setup_texture(int mu_steps, int nu_steps, int current_integral, double *host__lb) {
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int2>();
  //create the array
  hipArray *cu_array;
  hipExtent size;
  size.width = 4;
  size.height = mu_steps;
  size.depth = nu_steps;
  cutilSafeCall(hipMalloc3DArray(&cu_array,&channelDesc,size));

  int2 *host_int2 = convert(host__lb, 4*mu_steps*nu_steps);
  
  //copy date to the array
  hipMemcpy3DParms copyParams = {0};
  copyParams.srcPtr   = make_hipPitchedPtr((void*)host_int2,
					    size.width*sizeof(int2), 
					    size.width, 
					    size.height);
  copyParams.dstArray = cu_array;
  copyParams.extent   = size;
  copyParams.kind     = hipMemcpyHostToDevice;
  cutilSafeCall( hipMemcpy3D(&copyParams) );
  cu_arrays[current_integral] = cu_array;
}

/**
   Allocates cu arrays for the tex_device_lb texture and sets up
   parts of the texture
*/
void allocate_cu_arrays(int number_integrals) {
  cu_arrays = (hipArray**) malloc(sizeof(hipArray*) * number_integrals);
  //set texture parameters
  tex_device_lb.normalized = false;
  tex_device_lb.filterMode = hipFilterModePoint;
  tex_device_lb.addressMode[0] = hipAddressModeClamp;
  tex_device_lb.addressMode[1] = hipAddressModeClamp;
}

void bind_texture(int current_integral) {
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int2>();
  printf("binding the tex_device_lb texture for integral %u\n",
	 current_integral);
  cutilSafeCall(hipBindTextureToArray(tex_device_lb, cu_arrays[current_integral], channelDesc));
}


template <unsigned int number_streams, unsigned int convolve> 
__global__ void gpu__integral_kernel3(	int in_step, int in_steps,
					double q, double r0,
					double *device__lb, double *device__V,
					double *background_integrals,
					double *stream_integrals) {
	int i, j, pos;

	double *shared__r_point = shared_mem;
	double *shared__qw_r3_N = &shared__r_point[convolve];

	//make it so thread 0 populates the shared memory
	//the old way relied on blockDim.x >= convolve
	//which is not always the case
	if (threadIdx.x == 0) {
	  for(unsigned int idx = 0;idx < convolve; ++idx) {
	        pos = (blockIdx.y * convolve * 2) + (idx * 2);

		shared__r_point[idx] = constant__r_constants[pos];
		shared__qw_r3_N[idx] = constant__r_constants[pos + 1];
	  }
	}

	double bg_int = 0.0;

	double *st_int = &shared__qw_r3_N[convolve];
	for (i = 0; i < number_streams; i++) {
	  st_int[i * blockDim.x + threadIdx.x] = 0.0;
	}

	__syncthreads();
//  	double sinb = tex3D(tex_device_lb, 0,
//  			   kernel3__mu_step, kernel3__nu_step);
//  	double sinl = tex3D(tex_device_lb, 1,
//  			   kernel3__mu_step, kernel3__nu_step);
//  	double cosb = tex3D(tex_device_lb, 2,
//  			   kernel3__mu_step, kernel3__nu_step);
//  	double cosl = tex3D(tex_device_lb, 3,
//  			   kernel3__mu_step, kernel3__nu_step);

	double dotted, sxyz0, sxyz1, sxyz2;	

	double zp, rs;
	double xyz0, xyz1, xyz2;
	
	double rg;

	for (i = 0; i < convolve; i++) {
	          xyz2 = shared__r_point[i] 
		    * tex3D_double(tex_device_lb, 0,
				   kernel3__mu_step, kernel3__nu_step);
		  zp = shared__r_point[i] 
		    * tex3D_double(tex_device_lb, 2,
				   kernel3__mu_step, kernel3__nu_step);
		  
		  xyz0 = zp * tex3D_double(tex_device_lb, 3,
					   kernel3__mu_step, kernel3__nu_step) - f_lbr_r;
		  
		  xyz1 = zp * tex3D_double(tex_device_lb, 1,
					   kernel3__mu_step, kernel3__nu_step);
		  
		  
		  rg = sqrt(xyz0*xyz0 + xyz1*xyz1 + ((xyz2*xyz2)/(q*q)));
		  rs = rg + r0;
		  
		  bg_int += shared__qw_r3_N[i] / (rg * rs * rs * rs);

		  for (j = 0; j < number_streams; j++) {
 			pos = (j * 3);
 			sxyz0 = xyz0 - constant__fstream_c[pos + 0];
 			sxyz1 = xyz1 - constant__fstream_c[pos + 1];
 			sxyz2 = xyz2 - constant__fstream_c[pos + 2];

 			dotted = constant__fstream_a[pos + 0] * sxyz0 
			  + constant__fstream_a[pos + 1] * sxyz1
			  + constant__fstream_a[pos + 2] * sxyz2;
			
 			sxyz0 -= dotted * constant__fstream_a[pos + 0];
 			sxyz1 -= dotted * constant__fstream_a[pos + 1];
 			sxyz2 -= dotted * constant__fstream_a[pos + 2];

 			pos = j * blockDim.x + threadIdx.x;
			
			st_int[pos] += (shared__qw_r3_N[i] 
				       * exp(-((sxyz0 * sxyz0) + (sxyz1 * sxyz1) + (sxyz2 * sxyz2)) /
					     constant__fstream_sigma_sq2[j]));
 		  }
	}
	
	//define V down here so that one to reduce the number of registers, because a register
	//will be reused
	double V = device__V[kernel3__r_step + (kernel3__r_steps * kernel3__nu_step)];
 	pos = threadIdx.x + (blockIdx.x * blockDim.x) + (blockIdx.y * gridDim.x * blockDim.x);

 	background_integrals[pos] += (bg_int * V);
 	for (i = 0; i < number_streams; i++) {
	        stream_integrals[pos] 
		  += (st_int[i * blockDim.x + threadIdx.x] * V);
		pos += (blockDim.x * gridDim.x * gridDim.y);
 	}
}
