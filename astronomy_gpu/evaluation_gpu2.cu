#include "hip/hip_runtime.h"
/*
Copyright 2008, 2009 Travis Desell, Dave Przybylo, Nathan Cole,
Boleslaw Szymanski, Heidi Newberg, Carlos Varela, Malik Magdon-Ismail
and Rensselaer Polytechnic Institute.

This file is part of Milkway@Home.

Milkyway@Home is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

Milkyway@Home is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with Milkyway@Home.  If not, see <http://www.gnu.org/licenses/>.
*/

extern "C++" {
#include "../astronomy/parameters.h"
#include "../astronomy/star_points.h"
#include "coords.h"
#include "cpu_coords.h"
#include "cpu_r_constants.h"
#include "r_constants.h"
#include "pi_constants.h"
#include "gauss_legendre.h"
}

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cutil_inline.h>


#define MAX_CONVOLVE 120
#define R_INCREMENT 20 


int	number_threads = 256;

int	mu_increment = 1;
int	wedge;
int	convolve;
int	number_streams;
int	number_integrals;

int	*r_steps;
int	*mu_steps;
int	*nu_steps;

int	*sizeof_V;
float	**device__V;		//V				-- float[nu][r]

int	*sizeof_r_constants;
float	**host__r_constants;

int	*sizeof_lb;
float	**device__lb;		//sinb, sinl, cosb, cosl	-- float[nu][mu][4]

int	*integral_size;
float	**host__background_integrals;
float	**host__stream_integrals;
float	**device__background_integrals;
float	**device__stream_integrals;

__device__ __constant__ float device__fstream_sigma_sq2[8];
__device__ __constant__ float device__fstream_a[12];
__device__ __constant__ float device__fstream_c[12];

__device__ __constant__ float device__dx[MAX_CONVOLVE];
__device__ __constant__ float device__qgaus_W[MAX_CONVOLVE];

__device__ __constant__ float device__background_weight[1];
__device__ __constant__ float device__stream_weight[4];

__device__ __constant__ float device__r_constants[MAX_CONVOLVE * 2 * R_INCREMENT];

int	number_stars;
float	*device__stars;

int	probability_size;
float	*device__probability;
float	*host__probability;

float	*device__reduce;
float	*host__reduce;

//extern "C" void gpu__initialize(ASTRONOMY_PARAMETERS *ap, STAR_POINTS *sp);
//extern "C" double gpu__likelihood(double *parameters);
//extern "C" void gpu__free_constants();


void gpu__initialize(	int ap_wedge, int ap_convolve, int ap_number_streams, int ap_number_integrals, 
			int *in__r_steps, double *r_min, double *r_step_size,
			int *in__mu_steps, double *mu_min, double *mu_step_size,
			int *in__nu_steps, double *nu_min, double *nu_step_size,
			int in__number_stars, double **stars) { 
	int i, j, pos;

	wedge = ap_wedge;
	convolve = ap_convolve;
	number_streams = ap_number_streams;
	number_integrals = ap_number_integrals;
//	printf("wedge: %d, convolve: %d, number_streams: %d, number_integrals: %d\n", wedge, convolve, number_streams, number_integrals);

	sizeof_V = (int*)malloc(number_integrals * sizeof(int));
	sizeof_r_constants = (int*)malloc(number_integrals * sizeof(int));
	sizeof_lb = (int*)malloc(number_integrals * sizeof(int));

	integral_size = (int*)malloc(number_integrals * sizeof(int));

	device__background_integrals = (float**)malloc(number_integrals * sizeof(float*));
	device__stream_integrals = (float**)malloc(number_integrals * sizeof(float*));
	host__background_integrals = (float**)malloc(number_integrals * sizeof(float*));
	host__stream_integrals = (float**)malloc(number_integrals * sizeof(float*));

	device__V = (float**)malloc(number_integrals * sizeof(float*));
	device__lb = (float**)malloc(number_integrals * sizeof(float*));
	host__r_constants = (float**)malloc(number_integrals * sizeof(float*));

//	printf("calculating integral constants\n");
	r_steps = (int*)malloc(number_integrals * sizeof(int));
	mu_steps = (int*)malloc(number_integrals * sizeof(int));
	nu_steps = (int*)malloc(number_integrals * sizeof(int));
	for (i = 0; i < number_integrals; i++) {
		r_steps[i] = in__r_steps[i];
		mu_steps[i] = in__mu_steps[i];
		nu_steps[i] = in__nu_steps[i];

		sizeof_V[i] = in__nu_steps[i] * in__r_steps[i];
		sizeof_r_constants[i] = in__r_steps[i] * convolve * 2;
		sizeof_lb[i] = in__mu_steps[i] * in__nu_steps[i] * 4;

		double *cpu__V, *cpu__r_const, *cpu__lb;
		cpu__gc_to_lb(wedge, mu_steps[i], mu_min[i], mu_step_size[i], nu_steps[i], nu_min[i], nu_step_size[i], &cpu__lb);
		cpu__r_constants(convolve, r_steps[i], r_min[i], r_step_size[i], mu_steps[i], mu_min[i], mu_step_size[i], nu_steps[i], nu_min[i], nu_step_size[i], &cpu__V, &cpu__r_const);

		float *host__V			= (float*)malloc(sizeof_V[i] * sizeof(float));
		float *host__lb			= (float*)malloc(sizeof_lb[i] * sizeof(float));
		host__r_constants[i] = (float*)malloc(sizeof_r_constants[i] * sizeof(float));

		long constants_size = 0;
		constants_size += sizeof_V[i] * sizeof(float);
		constants_size += sizeof_r_constants[i] * sizeof(float); 
		constants_size += sizeof_lb[i] * sizeof(float);

//		printf("sizeof_V[%d]: %d\n", i, sizeof_V[i] * sizeof(float));
//		printf("sizeof_r_constants[%d]: %d\n", i, sizeof_r_constants[i] * sizeof(float));
//		printf("sizeof_lb[%d]: %d\n", i, sizeof_lb[i] * sizeof(float));

//		printf("Allocating %ld bytes for constants on GPU.\n", constants_size);

		for (j = 0; j < sizeof_V[i]; j++) {
			host__V[j] = (float)cpu__V[j];
		}
		for (j = 0; j < sizeof_r_constants[i]; j++) {
			host__r_constants[i][j] = (float)cpu__r_const[j];
		}
		for (j = 0; j < sizeof_lb[i]; j++) {
			host__lb[j] = (float)cpu__lb[j];
		}

//		printf("freeing cpu constants\n");
		free(cpu__V);
		free(cpu__r_const);
		free(cpu__lb);

//		printf("device malloc\n");

		cutilSafeCall( hipMalloc((void**) &(device__V[i]), sizeof_V[i] * sizeof(float)) );
		cutilSafeCall( hipMalloc((void**) &(device__lb[i]), sizeof_lb[i] * sizeof(float)) );

//		printf("device memcpy\n");

		cutilSafeCall( hipMemcpy(device__V[i], host__V, sizeof_V[i] * sizeof(float), hipMemcpyHostToDevice) );
		cutilSafeCall( hipMemcpy(device__lb[i], host__lb, sizeof_lb[i] * sizeof(float), hipMemcpyHostToDevice) );

		free(host__V);
		free(host__lb);

		integral_size[i] = R_INCREMENT * in__nu_steps[i] * in__mu_steps[i];
//		printf("Allocating %d bytes for integral data on GPU\n", (number_streams + 1) * integral_size[i] * sizeof(float));

		cutilSafeCall( hipMalloc((void**) &device__background_integrals[i], integral_size[i] * sizeof(float)) );
		cutilSafeCall( hipMalloc((void**) &device__stream_integrals[i], number_streams * integral_size[i] * sizeof(float)) );
		host__background_integrals[i] = (float*)malloc(integral_size[i] * sizeof(float));
		host__stream_integrals[i] = (float*)malloc(number_streams * integral_size[i] * sizeof(float));
	}

	cutilSafeCall( hipMalloc((void**) &device__reduce, 64 * sizeof(float)) );
	host__reduce = (float*)malloc(64 * sizeof(float));

//	printf("initializing constants for %d stars\n", number_stars);

	number_stars = in__number_stars;
	float *host__stars = (float*)malloc(number_stars * 5 * sizeof(float));
	for (i = 0; i < number_stars; i++) {
		pos = i * 5;
		host__stars[pos] = (float)sin(stars[i][1] * D_DEG2RAD);
		host__stars[pos + 1] = (float)sin(stars[i][0] * D_DEG2RAD);
		host__stars[pos + 2] = (float)cos(stars[i][1] * D_DEG2RAD);
		host__stars[pos + 3] = (float)cos(stars[i][0] * D_DEG2RAD);
		host__stars[pos + 4] = (float)stars[i][2];
	}
//	printf("allocating %d bytes for device__stars\n", number_stars * 5 * sizeof(float));
	cutilSafeCall( hipMalloc((void**) &device__stars, number_stars * 5 * sizeof(float)) );
	cutilSafeCall( hipMemcpy(device__stars, host__stars, number_stars * 5 * sizeof(float), hipMemcpyHostToDevice) );

	free(host__stars);

//	printf("initializing qgaus constants\n");

	double *d_qgaus_W = (double*)malloc(convolve * sizeof(double));
	double *d_qgaus_X = (double*)malloc(convolve * sizeof(double));

	d_gauss_legendre(-1.0, 1.0, d_qgaus_X, d_qgaus_W, convolve);
	float *host__dx = (float*)malloc(convolve * sizeof(float));
	float *host__qgaus_W = (float*)malloc(convolve * sizeof(float));
	for (i = 0; i < convolve; i++) {
		host__dx[i] = (float)(3.0 * d_stdev * d_qgaus_X[i]);
		host__qgaus_W[i] = (float)d_qgaus_W[i];
	}
	free(d_qgaus_W);
	free(d_qgaus_X);

	cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(device__dx), host__dx, convolve * sizeof(float), 0, hipMemcpyHostToDevice) );
	cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(device__qgaus_W), host__qgaus_W, convolve * sizeof(float), 0, hipMemcpyHostToDevice) );

	//	free(host__dx);
	//	free(host__qgaus_W);

//	printf("mallocing GPU bg and stream probability: %d bytes\n", number_threads * sizeof(float));

	probability_size = number_threads;
	cutilSafeCall( hipMalloc((void**) &device__probability, probability_size * sizeof(float)) );

//	printf("mallocing host bg and stream probability\n");

	host__probability = (float*)malloc(probability_size * sizeof(float));
}

void gpu__free_constants() {
	int i;
	for (i = 0; i < number_integrals; i++) {
		cutilSafeCall( hipFree(device__V[i]) );
		cutilSafeCall( hipFree(device__lb[i]) );
		cutilSafeCall( hipFree(device__background_integrals[i]) );
		cutilSafeCall( hipFree(device__stream_integrals[i]) );
		free(host__background_integrals[i]);
		free(host__stream_integrals[i]);
		free(host__r_constants[i]);
	}

	cutilSafeCall( hipFree(device__stars) );

	free(host__background_integrals);
	free(host__stream_integrals);
	free(device__V);
	free(host__r_constants);
	free(device__lb);
	free(device__background_integrals);
	free(device__stream_integrals);

	free(r_steps);
	free(mu_steps);
	free(nu_steps);

	free(sizeof_V);
	free(sizeof_lb);
	free(sizeof_r_constants);
	free(integral_size);

	cutilSafeCall( hipFree(device__probability) );
	free(host__probability);

	cutilSafeCall( hipFree(device__reduce) );
	free(host__reduce);
}

template <unsigned int number_streams>
__global__ void gpu__zero_integrals(float *background_integrals, float *stream_integrals) {
	int pos = threadIdx.x + (blockIdx.x * blockDim.x) + (blockIdx.y * blockDim.x * gridDim.x);

	background_integrals[pos] = 0;
	for (int i = 0; i < number_streams; i++) stream_integrals[(i * gridDim.y * gridDim.x * blockDim.x) + pos] = 0;
}

#define kernel3__mu_step	blockIdx.x
#define kernel3__mu_steps	gridDim.x
#define kernel3__r_step		(in_step + blockIdx.y)
#define kernel3__r_steps	in_steps
#define kernel3__nu_step	threadIdx.x
#define kernel3__nu_steps	blockDim.x

template <unsigned int number_streams> 
__global__ void gpu__integral_kernel3(	int convolve, int in_step, int in_steps,
		float q, float r0,
		float *device__lb, float *device__V,
		float *background_integrals, float *stream_integrals) {
	int i, j;

	float V = device__V[kernel3__r_step + (kernel3__r_steps * kernel3__nu_step)];

	int pos = ((kernel3__nu_step * kernel3__mu_steps) + kernel3__mu_step) * 4; 
	float sinb = device__lb[pos];
	float sinl = device__lb[pos + 1];
	float cosb = device__lb[pos + 2];
	float cosl = device__lb[pos + 3];

	float rg, xyz0, xyz1, xyz2;
	float dotted, sxyz0, sxyz1, sxyz2;

	float bg_int = 0.0;
	float st_int[number_streams];
	for (i = 0; i < number_streams; i++) st_int[i] = 0;

	float r_point, qw_r3_N;
	float zp, rs;
	for (i = 0; i < convolve; i++) {
		pos = (blockIdx.y * convolve * 2) + (i * 2);
		r_point = device__r_constants[pos];
		qw_r3_N = device__r_constants[pos + 1];

		xyz2 = r_point * sinb;
		zp = r_point * cosb;
		xyz0 = zp * cosl - f_lbr_r;
		xyz1 = zp * sinl;

		rg = sqrt(xyz0*xyz0 + xyz1*xyz1 + (xyz2*xyz2)/(q*q));
		rs = rg + r0;
		bg_int += qw_r3_N / (rg * rs * rs * rs);

		for (j = 0; j < number_streams; j++) {
			pos = (j * 3);
			sxyz0 = xyz0 - device__fstream_c[pos];
			sxyz1 = xyz1 - device__fstream_c[pos + 1];
			sxyz2 = xyz2 - device__fstream_c[pos + 2];

			dotted = device__fstream_a[pos] * sxyz0 + device__fstream_a[pos + 1] * sxyz1 + device__fstream_a[pos + 2] * sxyz2;

			sxyz0 -= dotted * device__fstream_a[pos];
			sxyz1 -= dotted * device__fstream_a[pos + 1];
			sxyz2 -= dotted * device__fstream_a[pos + 2];

			st_int[j] += qw_r3_N * exp(-((sxyz0 * sxyz0) + (sxyz1 * sxyz1) + (sxyz2 * sxyz2)) / device__fstream_sigma_sq2[j]);
		}
	}
	
	pos = threadIdx.x + (blockIdx.x * blockDim.x) + (blockIdx.y * gridDim.x * blockDim.x);
	background_integrals[pos] += bg_int * V;
	for (i = 0; i < number_streams; i++) stream_integrals[pos + (blockDim.x * gridDim.x * gridDim.y * i)] += st_int[i] * V;
}

void cpu__sum_integrals(int iteration, double *background_integral, double *stream_integrals) {
	int i, j;

	cutilSafeCall( hipMemcpy(host__background_integrals[iteration], device__background_integrals[iteration], integral_size[iteration] * sizeof(float), hipMemcpyDeviceToHost) );

	double sum = 0.0;
	for (i = 0; i < integral_size[iteration]; i++) {
		sum += (double)(host__background_integrals[iteration][i]);
//		printf("background_integral[%d/%d]: %.15f\n", i, integral_size[iteration], host__background_integrals[iteration][i]);
	}
	if (iteration == 0) *background_integral = sum;
	else *background_integral -= sum;

	cutilSafeCall( hipMemcpy(host__stream_integrals[iteration], device__stream_integrals[iteration], number_streams * integral_size[iteration] * sizeof(float), hipMemcpyDeviceToHost) );
	for (i = 0; i < number_streams; i++) {
		sum = 0.0;
		for (j = 0; j < integral_size[iteration]; j++) {
			sum += (double)(host__stream_integrals[iteration][j + (i * integral_size[iteration])]);
//			printf("stream_integral: %.15f\n", host__stream_integrals[iteration][j + (i * integral_size[iteration])]);
		}
		if (iteration == 0) stream_integrals[i] = sum;
		else stream_integrals[i] -= sum;
	}
}

/********
 *	Likelihood calculation
 ********/

template <unsigned int number_streams>
__global__ void gpu__zero_likelihood(int block_size, float *device__probability) {
	device__probability[threadIdx.x] = 0;
}

void cpu__sum_likelihood(int block_size, double *probability) {
	int i;

	cutilSafeCall( hipMemcpy(host__probability, device__probability, probability_size * sizeof(float), hipMemcpyDeviceToHost) );

//	*probability = 0.0;

	for (i = 0; i < block_size; i++) {
		*probability += host__probability[i];
//		printf("bg_prob: %.15f\n", host__probability[i]);
	}
}

template <unsigned int number_streams>
__global__ void gpu__likelihood_kernel(	int offset, int convolve,
		float q, float r0,
		float coeff, 
		float *device__stars,
		float *probability) {
	int i;
	int pos = (offset + threadIdx.x) * 5;
	float sinb = device__stars[pos];
	float sinl = device__stars[pos + 1];
	float cosb = device__stars[pos + 2];
	float cosl = device__stars[pos + 3];
	float coords = device__stars[pos + 4];

	float rg, xyz0, xyz1, xyz2;
	float dotted, sxyz0, sxyz1, sxyz2;

	float bg_int = 0.0;
	float st_int[number_streams];
	for (i = 0; i < number_streams; i++) st_int[i] = 0.0;

	float gPrime = 5.0f * (log10(coords * 1000.0f) - 1.0f) + f_absm;
	float exponent = exp(sigmoid_curve_1 * (gPrime - sigmoid_curve_2));
	float reff_value = sigmoid_curve_0 / (exponent + 1);
	float rPrime3 = coords * coords * coords;

	float reff_xr_rp3 = reff_value * f_xr / rPrime3;

	float r_point, qw_r3_N;
	float zp, rs, g;

	for (i = 0; i < convolve; i++) {
		g = gPrime + device__dx[i];

		r_point = pow(10.0f, (g - f_absm)/5.0f + 1.0f) / 1000.0f;
		rPrime3 = r_point * r_point * r_point;

		qw_r3_N = device__qgaus_W[i] * rPrime3 * coeff * exp( -((g - gPrime) * (g - gPrime) / (2 * f_stdev * f_stdev)) );

		xyz2 = r_point * sinb;
		zp = r_point * cosb;
		xyz0 = zp * cosl - f_lbr_r;
		xyz1 = zp * sinl;

		rg = sqrt(xyz0*xyz0 + xyz1*xyz1 + (xyz2*xyz2)/(q*q));
		rs = rg + r0;

		bg_int += qw_r3_N / (rg * rs * rs * rs);

		for (int j = 0; j < number_streams; j++) {
			pos = (j * 3);
			sxyz0 = xyz0 - device__fstream_c[pos];
			sxyz1 = xyz1 - device__fstream_c[pos + 1];
			sxyz2 = xyz2 - device__fstream_c[pos + 2];

			dotted = device__fstream_a[pos] * sxyz0 + device__fstream_a[pos + 1] * sxyz1 + device__fstream_a[pos + 2] * sxyz2;

			sxyz0 -= dotted * device__fstream_a[pos];
			sxyz1 -= dotted * device__fstream_a[pos + 1];
			sxyz2 -= dotted * device__fstream_a[pos + 2];

			st_int[j] += qw_r3_N * exp(-((sxyz0 * sxyz0) + (sxyz1 * sxyz1) + (sxyz2 * sxyz2)) / device__fstream_sigma_sq2[j]);
		}
	}
	float probability_sum = 0.0;
	probability_sum += bg_int * reff_xr_rp3 * device__background_weight[0];
	for (i = 0; i < number_streams; i++) {
		probability_sum += st_int[i] * reff_xr_rp3 * device__stream_weight[i];
	}
//	printf("bg_prob %.15f st_prob[0]: %.15f st_prob[1]: %.15f, prob_sum: %.15f\n", (bg_int * reff_xr_rp3), (st_int[0] * reff_xr_rp3), (st_int[1] * reff_xr_rp3), probability_sum);

	if (probability_sum == 0.0) probability[threadIdx.x] += -238.0;
	else probability[threadIdx.x] += log(probability_sum)/log(10.0);
}

/********
 *	Run the GPU kernels and get the probability
 ********/

#define stream_parameters(x, y) parameters[(x * 6) + y + 3]
#define stream_weights(x) parameters[(x * 6) + 2]
//#define background_weight parameters[0]
#define background_weight 0.0
//#define alpha parameters[1]
#define q parameters[0]
#define r0 parameters[1]
//#define delta parameters[4]

double gpu__likelihood(double *parameters) {
	int i, j;

	double stream_c[3], lbr[3];
	float fstream_a[number_streams * 3], fstream_c[number_streams * 3], fstream_sigma_sq2[number_streams];

	for (i = 0; i < number_streams; i++) {
		fstream_sigma_sq2[i] = (float)(2.0 * stream_parameters(i,4) * stream_parameters(i,4));

		fstream_a[(i * 3)] = (float)( sin(stream_parameters(i,2)) * cos(stream_parameters(i,3)) );
		fstream_a[(i * 3) + 1] = (float)( sin(stream_parameters(i,2)) * sin(stream_parameters(i,3)) );
		fstream_a[(i * 3) + 2] = (float)( cos(stream_parameters(i,2)) );

		gc_to_gal(wedge, stream_parameters(i,0) * D_DEG2RAD, 0 * D_DEG2RAD, &(lbr[0]), &(lbr[1]));
		lbr[2] = stream_parameters(i,1);
		d_lbr2xyz(lbr, stream_c);

		fstream_c[(i * 3)] = (float)stream_c[0]; 
		fstream_c[(i * 3) + 1] = (float)stream_c[1];
		fstream_c[(i * 3) + 2] = (float)stream_c[2];
	}

	cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(device__fstream_sigma_sq2), fstream_sigma_sq2, number_streams * sizeof(float), 0, hipMemcpyHostToDevice) ); 
	cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(device__fstream_a), fstream_a, number_streams * 3 * sizeof(float), 0, hipMemcpyHostToDevice) );
	cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(device__fstream_c), fstream_c, number_streams * 3 * sizeof(float), 0, hipMemcpyHostToDevice) );

	double background_integral = 0.0;
	double *stream_integrals = (double*)malloc(number_streams * sizeof(double));
	for (i = 0; i < number_streams; i++) stream_integrals[i] = 0.0;

	double coeff = 1.0 / (d_stdev * sqrt(2.0 * D_PI));

	for (i = 0; i < number_integrals; i++) {
		dim3 dimGrid(mu_steps[i], R_INCREMENT);

		gpu__zero_integrals<2><<<dimGrid, nu_steps[i]>>>(device__background_integrals[i], device__stream_integrals[i]);
		for (j = 0; j < r_steps[i]; j += R_INCREMENT) {
			cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(device__r_constants), &(host__r_constants[i][j * convolve * 2]), R_INCREMENT * convolve * 2 * sizeof(float), 0, hipMemcpyHostToDevice) );

			switch(number_streams) {
				case 1:	gpu__integral_kernel3<1><<<dimGrid, nu_steps[i]>>>(	convolve, j, r_steps[i], 
							q, r0,
							device__lb[i], device__V[i], 
							device__background_integrals[i],
							device__stream_integrals[i]);
					break;
				case 2:	gpu__integral_kernel3<2><<<dimGrid, nu_steps[i]>>>(	convolve, j, r_steps[i], 
							q, r0,
							device__lb[i], device__V[i], 
							device__background_integrals[i],
							device__stream_integrals[i]);
					break;
				case 3:	gpu__integral_kernel3<3><<<dimGrid, nu_steps[i]>>>(	convolve, j, r_steps[i], 
							q, r0,
							device__lb[i], device__V[i], 
							device__background_integrals[i],
							device__stream_integrals[i]);
					break;
				case 4:	gpu__integral_kernel3<4><<<dimGrid, nu_steps[i]>>>(	convolve, j, r_steps[i], 
							q, r0,
							device__lb[i], device__V[i], 
							device__background_integrals[i],
							device__stream_integrals[i]);
					break;
			}
//			cpu__sum_integrals(i, &background_integral, stream_integrals);
//			printf("background_integral: %.15lf, stream_integral[0]: %.15lf, stream_integral[1]: %.15lf\n", background_integral, stream_integrals[0], stream_integrals[1]);
		}
		cpu__sum_integrals(i, &background_integral, stream_integrals);
//		printf("background_integral: %.15lf, stream_integral[0]: %.15lf, stream_integral[1]: %.15lf\n", background_integral, stream_integrals[0], stream_integrals[1]);
	}

	int block_size;

	double *stream_weight = (double*)malloc(number_streams * sizeof(double));
	double exp_weight = exp(background_weight);
	double sum_exp_weights = exp_weight; 
	double bg_weight = exp_weight/background_integral;
	for (i = 0; i < number_streams; i++) {
		exp_weight = exp(stream_weights(i));
		sum_exp_weights += exp_weight;
		stream_weight[i] = exp_weight/stream_integrals[i];
	}

	float f_background_weight[1];
	float *f_stream_weight = (float*)malloc(number_streams * sizeof(float));
	f_background_weight[0] = (float)( bg_weight / sum_exp_weights );
	for (i = 0; i < number_streams; i++) {
		f_stream_weight[i] = (float)( stream_weight[i] / sum_exp_weights );
	}

	cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(device__background_weight), f_background_weight, 1 * sizeof(float), 0, hipMemcpyHostToDevice) );
	cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(device__stream_weight), f_stream_weight, number_streams * sizeof(float), 0, hipMemcpyHostToDevice) );

	double likelihood = 0.0;
	gpu__zero_likelihood<2><<<1, number_threads>>>(number_threads, device__probability);
	for (i = 0; i < number_stars; i += number_threads) {
		block_size = min(number_threads, number_stars - i);
		switch (number_streams) {
			case 1:	gpu__likelihood_kernel<1><<<1, block_size>>>(	i, convolve,
										q, r0,
										(float)coeff,
										device__stars,
										device__probability);
			break;
			case 2:	gpu__likelihood_kernel<2><<<1, block_size>>>(	i, convolve,
										q, r0,
										(float)coeff,
										device__stars,
										device__probability);
			break;
			case 3:	gpu__likelihood_kernel<3><<<1, block_size>>>(	i, convolve,
										q, r0,
										(float)coeff,
										device__stars,
										device__probability);
			break;
			case 4:	gpu__likelihood_kernel<4><<<1, block_size>>>(	i, convolve,
										q, r0,
										(float)coeff,
										device__stars,
										device__probability);
			break;
		}
	}
	cpu__sum_likelihood(number_threads, &likelihood);
	likelihood /= number_stars;
//	printf("likelihood: %.15lf\n", likelihood);
	return likelihood;
}
