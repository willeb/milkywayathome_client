#include "hip/hip_runtime.h"
/*
Copyright 2008, 2009 Travis Desell, Dave Przybylo, Nathan Cole,
Boleslaw Szymanski, Heidi Newberg, Carlos Varela, Malik Magdon-Ismail
and Rensselaer Polytechnic Institute.

This file is part of Milkway@Home.

Milkyway@Home is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

Milkyway@Home is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with Milkyway@Home.  If not, see <http://www.gnu.org/licenses/>.
*/

texture<float, 3, hipReadModeElementType> tex_device_lb;

/**
   Sets up a 3D texture for device__lb, right now it only
   support 1 integral.  In the future it should support
   more by copying from device memory the next integral.
 */
void setup_texture(int mu_steps, int nu_steps, float *host__lb) {
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
  //create the array
  hipArray *cu_array;
  hipExtent size;
  size.width = 4;
  size.height = mu_steps;
  size.depth = nu_steps;
  cutilSafeCall(hipMalloc3DArray(&cu_array,&channelDesc,size));
  
  //copy date to the array
  hipMemcpy3DParms copyParams = {0};
  copyParams.srcPtr   = make_hipPitchedPtr((void*)host__lb,
					    size.width*sizeof(float), 
					    size.width, 
					    size.height);
  copyParams.dstArray = cu_array;
  copyParams.extent   = size;
  copyParams.kind     = hipMemcpyHostToDevice;
  cutilSafeCall( hipMemcpy3D(&copyParams) );
  
  //set texture parameters
  tex_device_lb.normalized = false;
  tex_device_lb.filterMode = hipFilterModePoint;
  tex_device_lb.addressMode[0] = hipAddressModeClamp;
  tex_device_lb.addressMode[1] = hipAddressModeClamp;
  
  //find the array to the texture
  cutilSafeCall(hipBindTextureToArray(tex_device_lb, cu_array, channelDesc));
}


template <unsigned int number_streams, unsigned int convolve> 
__global__ void gpu__integral_kernel3(	int in_step, int in_steps,
		float q, float r0,
		float *device__lb, float *device__V,
		float *background_integrals, float *background_correction, 
		float *stream_integrals, float *stream_correction) {
	int i, j, pos;

	float *shared__r_point = shared_mem;
	float *shared__qw_r3_N = &shared__r_point[convolve];

	if (threadIdx.x < convolve) {
		pos = (blockIdx.y * convolve * 2) + (threadIdx.x * 2);

		shared__r_point[threadIdx.x] = constant__r_constants[pos];
		shared__qw_r3_N[threadIdx.x] = constant__r_constants[pos + 1];
	}

	float bg_int, bg_int_correction;
	bg_int = 0.0;
	bg_int_correction = 0.0; 

	float *st_int = &shared__qw_r3_N[convolve];
	float *st_int_correction = &st_int[blockDim.x * number_streams];
	for (i = 0; i < number_streams; i++) {
	  st_int[i * blockDim.x + threadIdx.x] = 0.0f;
	  st_int_correction[i * blockDim.x  + threadIdx.x] = 0.0f;
	}

	__syncthreads();

	float corrected_next_term, new_sum;
	float dotted, sxyz0, sxyz1, sxyz2;

//  	float sinb = tex3D(tex_device_lb, 0,
//  			   kernel3__mu_step, kernel3__nu_step);
//  	float sinl = tex3D(tex_device_lb, 1,
//  			   kernel3__mu_step, kernel3__nu_step);
//  	float cosb = tex3D(tex_device_lb, 2,
//  			   kernel3__mu_step, kernel3__nu_step);
//  	float cosl = tex3D(tex_device_lb, 3,
//  			   kernel3__mu_step, kernel3__nu_step);
	
	float zp, rs;
	float xyz0, xyz1, xyz2;
	
	float rg;

	for (i = 0; i < convolve; i++) {
		  xyz2 = shared__r_point[i] * tex3D(tex_device_lb, 0,
						    kernel3__mu_step, kernel3__nu_step);
		  zp = shared__r_point[i] * tex3D(tex_device_lb, 2,
						  kernel3__mu_step, kernel3__nu_step);
		  xyz0 = zp * tex3D(tex_device_lb, 3,
				    kernel3__mu_step, kernel3__nu_step) - f_lbr_r;
		  xyz1 = zp * tex3D(tex_device_lb, 1,
				    kernel3__mu_step, kernel3__nu_step);
		  
		  //__fdividef providers faster fp division, with restrictions on
		  // the fact that (q*q) < 2^126 (appendix b.2.1 in nvidia programming guide)
		  rg = sqrtf(xyz0*xyz0 + xyz1*xyz1 + __fdividef((xyz2*xyz2),(q*q)));
		  rs = rg + r0;
		  

		  corrected_next_term = shared__qw_r3_N[i] / (rg * rs * rs * rs) - bg_int_correction;
		  new_sum = bg_int + corrected_next_term;
		  bg_int_correction = (new_sum - bg_int) - corrected_next_term;
		  bg_int = new_sum;
		  for (j = 0; j < number_streams; j++) {
			pos = (j * 3);
			sxyz0 = xyz0 - constant__fstream_c[pos];
			sxyz1 = xyz1 - constant__fstream_c[pos + 1];
			sxyz2 = xyz2 - constant__fstream_c[pos + 2];

			dotted = constant__fstream_a[pos] * sxyz0 + constant__fstream_a[pos + 1] * sxyz1 + constant__fstream_a[pos + 2] * sxyz2;

			sxyz0 -= dotted * constant__fstream_a[pos];
			sxyz1 -= dotted * constant__fstream_a[pos + 1];
			sxyz2 -= dotted * constant__fstream_a[pos + 2];

			corrected_next_term = (shared__qw_r3_N[i] * exp(-((sxyz0 * sxyz0) + (sxyz1 * sxyz1) + (sxyz2 * sxyz2)) / constant__fstream_sigma_sq2[j])) - st_int_correction[j * blockDim.x  + threadIdx.x];
			new_sum = st_int[j * blockDim.x  + threadIdx.x] + corrected_next_term;
			st_int_correction[j * blockDim.x  + threadIdx.x] = (new_sum - st_int[j * blockDim.x  + threadIdx.x]) - corrected_next_term;
			st_int[j * blockDim.x + threadIdx.x] = new_sum;
		  }
	}
	
	//define V down here so that one to reduce the number of registers, because a register
	//will be reused
	float V = device__V[kernel3__r_step + (kernel3__r_steps * kernel3__nu_step)];
	pos = threadIdx.x + (blockIdx.x * blockDim.x) + (blockIdx.y * gridDim.x * blockDim.x);

	corrected_next_term = (bg_int * V) - background_correction[pos];
	new_sum = background_integrals[pos] + corrected_next_term;
	background_correction[pos] = (new_sum - background_integrals[pos]) - corrected_next_term;
	background_integrals[pos] = new_sum;
	for (i = 0; i < number_streams; i++) {
		corrected_next_term = (st_int[i * blockDim.x + threadIdx.x] * V) - stream_correction[pos];
		new_sum = stream_integrals[pos] + corrected_next_term;
		stream_correction[pos] = (new_sum - stream_integrals[pos]) - corrected_next_term;
		stream_integrals[pos] = new_sum;

		pos += (blockDim.x * gridDim.x * gridDim.y);
	}
}
